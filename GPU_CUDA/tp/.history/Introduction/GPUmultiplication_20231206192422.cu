#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

// Kernel function to add the elements of two matrices
__global__ void add(int rows, int cols, float *matA, float *matB, float *matC)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols)
    {
        int index = row * cols + col;
        matC[index] = matA[index] + matB[index];
    }
}

// kernel function to multiply two matrices
__global__ void multiply(int rows, int cols, float *matA, float *matB, float *matC)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols)
    {
        int index = row * cols + col;
        for (int k = 0; k < cols; k++)
        {
            matC[index] += matA[row * cols + k] * matB[k * cols + col];
        }
    }
}

int main(void)
{
    int rows = 1024; // number of rows
    int cols = 1024; // number of columns
    int N = rows * cols;

    float *matA, *matB, *matC, *d_matA, *d_matB, *d_matC;
    matA = (float *)malloc(N * sizeof(float));
    matB = (float *)malloc(N * sizeof(float));
    matC = (float *)malloc(N * sizeof(float));

    hipMalloc(&d_matA, N * sizeof(float));
    hipMalloc(&d_matB, N * sizeof(float));
    hipMalloc(&d_matC, N * sizeof(float));

    // Initialize matA and matB arrays on the host
    for (int i = 0; i < N; i++)
    {
        matA[i] = 1.0f;
        matB[i] = 2.0f;
    }
    // time start
    clock_t start = clock();
    // Copy the arrays to the device[GPU]
    hipMemcpy(d_matA, matA, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_matB, matB, N * sizeof(float), hipMemcpyHostToDevice);

    // Perform addition on the GPU
    dim3 blockSize(16, 16);                       // number of threads per block
    dim3 numBlocks(ceil((float)(cols)) / blockSize.x,  
                   ceil((float)(rows)) / blockSize.y);  // number of blocks
    multiply<<<numBlocks, blockSize>>>(rows, cols, d_matA, d_matB, d_matC); // call multiply kernel function

    // Copy the result back to the host
    hipMemcpy(matC, d_matC, N * sizeof(float), hipMemcpyDeviceToHost);
    clock_t end = clock(); // time end
    double time_spent = (double)(end - start) / CLOCKS_PER_SEC;
    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(matC[i] - 2048.0f));
    // print matA and matB and matC
    for (int i = 0; i < 10; i++)
    {
        printf("[GPU] matA[%d] = %f\n", i, matA[i]);
    }
    for (int i = 0; i < 10; i++)
    {
        printf("[GPU] matB[%d] = %f\n", i, matB[i]);
    }
    for (int i = 0; i < 10; i++)
    {
        printf("[GPU] matC[%d] = %f\n", i, matC[i]);
    }
    printf("[GPU] Max error: %f\n", maxError);
    printf("[GPU] Time spent: %f\n", time_spent);
    // Free memory
    hipFree(d_matA);
    hipFree(d_matB);
    hipFree(d_matC);
    free(matA);
    free(matB);
    free(matC);

    return 0;
}
