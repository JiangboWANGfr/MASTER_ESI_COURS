/**
 * @file GPUmultiplication.cu
 * @author jiangboWANG(jiangbo.wang@ensea.fr)
 * @brief  GPU multiplication of two matrices of size 1024*1024 using CUDA
 *          we campare the time spent between CPU and GPU(see CPUmultiplication.c)
 * @version 0.1
 * @date 2023-12-06
 * 
 * @copyright Copyright (c) 2023
 * 
 */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>


/**
 * @brief kernel function to multiply two matrices
 * @details in the GPU, we use the grid of blocks and threads to compute the 
 *          multiplication of two matrices ,so we just need to use one for loop.
 *          1. we calculate the row and column indices of each thread, this is done using 
 *             the block index (blockIdx.x and blockIdx.y), block dimension (blockDim.x 
 *             and blockDim.y), and thread index within the block (threadIdx.x and threadIdx.y). 
 * 
 * 
 * @param rows number of rows
 * @param cols number of columns
 * @param matA matrix A
 * @param matB matrix B
 * @param matC matrix C = matA * matB
*/
__global__ void multiply(int rows, int cols, float *matA, float *matB, float *matC)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols)
    {
        int index = row * cols + col;
        for (int k = 0; k < cols; k++)
        {
            matC[index] += matA[row * cols + k] * matB[k * cols + col];
        }
    }
}

/**
 * @brief main function
 *        0. allocate memory on the host and device
 *        1. initialize matA and matB on the host 
 *        2. copy matA and matB from the host to the device[GPU]
 *        3. define the number of threads per block and the number of blocks
 *        4. call the kernel function
 *        5. copy the result from the device to the host
 *        6. print the result
 *        7. free memory on the host and device
 * @note the time spent is 0.025s using GPU ,compare to 98.56s using CPU
 * @return int 
*/
int main(void)
{
    int rows = 1024; // number of rows
    int cols = 1024; // number of columns
    int N = rows * cols;

    float *matA, *matB, *matC, *d_matA, *d_matB, *d_matC;
    matA = (float *)malloc(N * sizeof(float));
    matB = (float *)malloc(N * sizeof(float));
    matC = (float *)malloc(N * sizeof(float));

    hipMalloc(&d_matA, N * sizeof(float));
    hipMalloc(&d_matB, N * sizeof(float));
    hipMalloc(&d_matC, N * sizeof(float));

    // Initialize matA and matB arrays on the host
    for (int i = 0; i < N; i++)
    {
        matA[i] = 1.0f;
        matB[i] = 2.0f;
    }
    // time start
    clock_t start = clock();
    // Copy the arrays to the device[GPU]
    hipMemcpy(d_matA, matA, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_matB, matB, N * sizeof(float), hipMemcpyHostToDevice);

    // Perform addition on the GPU
    dim3 blockSize(16, 16);                       // number of threads per block
    dim3 numBlocks(ceil((float)(cols)) / blockSize.x,  
                   ceil((float)(rows)) / blockSize.y);  // number of blocks
    multiply<<<numBlocks, blockSize>>>(rows, cols, d_matA, d_matB, d_matC); // call multiply kernel function

    // Copy the result back to the host
    hipMemcpy(matC, d_matC, N * sizeof(float), hipMemcpyDeviceToHost);
    clock_t end = clock(); // time end
    double time_spent = (double)(end - start) / CLOCKS_PER_SEC; // calculate time spent
    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(matC[i] - 2048.0f));
    // print matA and matB and matC
    for (int i = 0; i < 10; i++)
    {
        printf("[GPU] matA[%d] = %f\n", i, matA[i]);
    }
    for (int i = 0; i < 10; i++)
    {
        printf("[GPU] matB[%d] = %f\n", i, matB[i]);
    }
    for (int i = 0; i < 10; i++)
    {
        printf("[GPU] matC[%d] = %f\n", i, matC[i]);
    }
    printf("[GPU] Max error: %f\n", maxError);
    printf("[GPU] Time spent: %f\n", time_spent);
    // Free memory
    hipFree(d_matA);
    hipFree(d_matB);
    hipFree(d_matC);
    free(matA);
    free(matB);
    free(matC);

    return 0;
}
