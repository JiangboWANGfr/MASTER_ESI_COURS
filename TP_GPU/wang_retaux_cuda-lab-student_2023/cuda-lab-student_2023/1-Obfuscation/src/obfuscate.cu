#include "hip/hip_runtime.h"
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include "obfuscate.hpp"

/************************************************************************************
                                        EXERCISE 1
*************************************************************************************/

/**
 * @brief This function should add the values of the pixels from two input buffers and store the result in an output buffer
 *        This function should be executed on the GPU, using CUDA
 * 
 * @param d_in_buffer_1 : input image 1, contains the values of the pixels
 * @param d_in_buffer_2 : input image 2, contains the values of the pixels  
 * @param width         : width of the image
 * @param height        : height of the image
 * @param d_out_buffer  : output image, contains the values of the pixels
 * @return __global__ 
 */
__global__ void k_cuda_exercise_1(uint8_t *d_in_buffer_1, uint8_t *d_in_buffer_2, uint32_t width, uint32_t height, uint8_t *d_out_buffer)
{
    // add an element coresponding to the thread index (hint: use threadIdx.x and threadIdx.y)

    // index_x and index_y are the indices of the element in the buffer
    int index_x = threadIdx.x + blockDim.x*blockIdx.x;
    int index_y = threadIdx.y + blockDim.y*blockIdx.y;
    if(index_x < width && index_y < height)
    {
        // this is the index of the element in the buffer
        int index = index_x + index_y * width;
        d_out_buffer[index] = d_in_buffer_1[index] + d_in_buffer_2[index];
    }
}

/**
 * @brief This function should add the values of the pixels from two input buffers and store the result in an output buffer
 * 
 * @param in_buffer_1  : input image 1, contains the values of the pixels,not RGB
 * @param in_buffer_2  : input image 2, contains the values of the pixels,not RGB
 * @param width        : width of the image
 * @param height       : height of the image
 * @param out_buffer   : output image, contains the values of the pixels
 */
void cuda_exercise_1(uint8_t *in_buffer_1, uint8_t *in_buffer_2, uint32_t width, uint32_t height, uint8_t *out_buffer)
{
    uint8_t *d_in_buffer_1 = NULL, *d_in_buffer_2 = NULL, *d_out_buffer = NULL;
    const int size_bytes = width * height * sizeof(uint8_t);
    std::cout << "width: " << width << " height: " << height << std::endl;


    hipEventRecord(start_memory);
    //allocate memory for d_in_buffer_1, d_in_buffer_2 and d_out_buffer
    hipMalloc(&d_in_buffer_1, size_bytes);
    hipMalloc(&d_in_buffer_2, size_bytes);
    hipMalloc(&d_out_buffer, size_bytes);
    //copy in_buffer_1 and in_buffer_2 to device
    hipMemcpy(d_in_buffer_1, in_buffer_1, size_bytes, hipMemcpyHostToDevice);    
    hipMemcpy(d_in_buffer_2, in_buffer_2, size_bytes, hipMemcpyHostToDevice);

    hipEventRecord(start_kernel);
    //launch kernels with the right block dimensions
    dim3 dimBlock(12,16);
    dim3 dimGrid(ceil(width/12.0), ceil(height/16.0));
    std::cout << "dimBlock: " << dimBlock.x << " " << dimBlock.y << std::endl;
    std::cout << "dimGrid: " << dimGrid.x << " " << dimGrid.y << std::endl;
    k_cuda_exercise_1<<<dimGrid, dimBlock>>>(d_in_buffer_1, d_in_buffer_2, width, height, d_out_buffer);

    hipEventRecord(start_copyback);
    //copy d_out_buffer back to host
    hipMemcpy(out_buffer, d_out_buffer, size_bytes, hipMemcpyDeviceToHost);

    //free all the device buffers
    hipFree(d_out_buffer);
    hipFree(d_in_buffer_1);
    hipFree(d_in_buffer_2);

    hipEventSynchronize(end);
}

/************************************************************************************
                                        EXERCISE 2
*************************************************************************************/

/**
 * @brief This function should add the R, G and B values of the pixels from two input buffers and store the result in an output buffer
 *        This function should be executed on the GPU, using CUDA
 *        So each thread should add the R, G and B values of a pixel from the two input buffers and store the result in the output buffer
 * 
 * @param d_in_buffer_1  : input image 1, contains the R, G and B values of the pixels
 * @param d_in_buffer_2  : input image 2, contains the R, G and B values of the pixels
 * @param width          : width of the image
 * @param height         : height of the image
 * @param d_out_buffer   : output image, contains the R, G and B values of the pixels
 * @return __global__ 
 */
__global__
void k_cuda_exercise_2(uint32_t *d_in_buffer_1, uint32_t *d_in_buffer_2, uint32_t width, uint32_t height, uint32_t *d_out_buffer)
{
    //add an element coresponding to the thread index (hint: use blockIdx.x, blockIdx.y, threadIdx.x and threadIdx.y)
    int index_x = threadIdx.x + blockDim.x * blockIdx.x;
    int index_y = threadIdx.y + blockDim.y * blockIdx.y;
    if(index_x <= width && index_y <= height)
    {
        // this is the index of the element in the buffer
        int index = index_x + index_y * width;
        // get the R, G and B values of the pixel from the two input buffers
        uint8_t R =  get_R(d_in_buffer_1[index]) + get_R(d_in_buffer_2[index]);
        uint8_t G =  get_G(d_in_buffer_1[index]) + get_G(d_in_buffer_2[index]);
        uint8_t B =  get_B(d_in_buffer_1[index]) + get_B(d_in_buffer_2[index]);
        // set the R, G and B values of the pixel in the output buffer
        d_out_buffer[index] = make_RGB(R, G, B);
    }
}


/**
 * @brief This function should add the R, G and B values of the pixels from two input buffers and store the result in an output buffer
 * 
 * @param in_buffer_1 : input image 1, contains the R, G and B values of the pixels
 * @param in_buffer_2 : input image 2, contains the R, G and B values of the pixels
 * @param width      : width of the image
 * @param height    : height of the image
 * @param out_buffer : output image, contains the R, G and B values of the pixels
 */
void cuda_exercise_2(uint32_t *in_buffer_1, uint32_t *in_buffer_2, uint32_t width, uint32_t height, uint32_t *out_buffer)
{
    uint32_t *d_in_buffer_1 = NULL, *d_in_buffer_2 = NULL, *d_out_buffer = NULL;
    const int size_bytes = width * height * sizeof(uint32_t);

    std::cout << "width: " << width << " height: " << height << std::endl;

    hipEventRecord(start_memory);
    // allocate memory for d_in_buffer_2 and d_out_buffer
    hipMalloc(&d_in_buffer_1, size_bytes);
    hipMalloc(&d_in_buffer_2, size_bytes);
    hipMalloc(&d_out_buffer, size_bytes);

    // copy in_buffer_2 to device
    hipMemcpy(d_in_buffer_1, in_buffer_1, size_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_in_buffer_2, in_buffer_2, size_bytes, hipMemcpyHostToDevice);

    hipEventRecord(start_kernel);
    //launch kernels with the right block and grid dimensions
    dim3 dimBlock(10, 10);
    dim3 dimGrid(ceil(width/10.0), ceil(height/10.0));
    std::cout << "dimBlock: " << dimBlock.x << " " << dimBlock.y << std::endl;
    std::cout << "dimGrid: " << dimGrid.x << " " << dimGrid.y << std::endl;
    k_cuda_exercise_2<<<dimGrid, dimBlock>>>(d_in_buffer_1, d_in_buffer_2, width, height, d_out_buffer);

    hipEventRecord(start_copyback);

    hipEventRecord(start_copyback);
    // copy d_out_buffer back to host
    hipMemcpy(out_buffer, d_out_buffer, size_bytes, hipMemcpyDeviceToHost);

    hipEventRecord(end);
    // free all the device buffers
    hipFree(d_in_buffer_1);
    hipFree(d_in_buffer_2);
    hipFree(d_out_buffer);

    hipEventSynchronize(end);
}